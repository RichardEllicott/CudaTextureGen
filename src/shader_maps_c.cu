#include "hip/hip_runtime.h"
#include "shader_maps_c.cuh"

namespace shader_maps_c {

#pragma region NORMAL_MAP

// Addressing helper
__device__ __forceinline__ int image_position_to_index(int x, int y, const int width, const int height, const bool wrap) {
    if (wrap) {
        x = (x % width + width) % width;
        y = (y % height + height) % height;
    } else {
        x = min(max(x, 0), width - 1);
        y = min(max(y, 0), height - 1);
    }
    return y * width + x;
}

// Normalize a 3D vector
__device__ __forceinline__ void normalize3(float &x, float &y, float &z) {
    float len = sqrtf(x * x + y * y + z * z);
    if (len > 1e-6f) {
        x /= len;
        y /= len;
        z /= len;
    }
}

__global__ void generate_normal_map_kernel(const float *__restrict__ heightmap,
                                           float *__restrict__ normalmap,
                                           int width, int height,
                                           float normal_scale, bool wrap) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height)
        return;

    // Offsets clockwise from top
    const int ox[8] = {0, 1, 1, 1, 0, -1, -1, -1};
    const int oy[8] = {-1, -1, 0, 1, 1, 1, 0, -1};

    float samples[8];
    for (int i = 0; i < 8; ++i) {
        int nx = x + ox[i];
        int ny = y + oy[i];
        int idx = image_position_to_index(nx, ny, width, height, wrap);
        samples[i] = heightmap[idx];
    }

    // Sobel operator
    float dx = (samples[1] + 2 * samples[2] + samples[3]) -
               (samples[7] + 2 * samples[6] + samples[5]);
    float dy = (samples[5] + 2 * samples[4] + samples[3]) -
               (samples[7] + 2 * samples[0] + samples[1]);

    float nx = dx * normal_scale;
    float ny = -dy * normal_scale;
    float nz = 1.0f;

    normalize3(nx, ny, nz);

    // Convert to [0,1] color space
    float r = 0.5f + 0.5f * nx;
    float g = 0.5f + 0.5f * ny;
    float b = 0.5f + 0.5f * nz;

    int base = (y * width + x) * 3;
    normalmap[base + 0] = r;
    normalmap[base + 1] = g;
    normalmap[base + 2] = b;
}

// host_in: pointer to heightmap data (width*height floats)
// host_out: pointer to output normal map (width*height*3 floats)
void ShaderMaps::generate_normal_map(
    const float *host_in, float *host_out,
    int width, int height,
    float scale, bool wrap) {

    size_t in_size = width * height * sizeof(float);
    size_t out_size = width * height * 3 * sizeof(float);

    float *d_in = nullptr;
    float *d_out = nullptr;

    hipMalloc(&d_in, in_size);
    hipMalloc(&d_out, out_size);

    hipMemcpy(d_in, host_in, in_size, hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x,
              (height + block.y - 1) / block.y);

    generate_normal_map_kernel<<<grid, block>>>(
        d_in, d_out,
        width, height,
        scale, wrap);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel launch failed: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(host_out, d_out, out_size, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
}

#pragma endregion

#pragma region AO_MAP

__global__ void generate_ao_map_kernel(
    const float *__restrict__ image, float *__restrict__ ao_map,
    int width, int height,
    int radius, bool wrap) {

    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height)
        return;

    const int base_index = y * width + x;
    const float base_height = image[base_index];

    float occlusion = 0.0f;
    float total_weight = 0.0f;

    // Accumulate within circular radius
    for (int dy = -radius; dy <= radius; ++dy) {
        for (int dx = -radius; dx <= radius; ++dx) {
            if (dx == 0 && dy == 0)
                continue;
            if (dx * dx + dy * dy > radius * radius)
                continue;

            const int sample_index = image_position_to_index(x + dx, y + dy, width, height, wrap);
            const float neighbor_height = image[sample_index];

            const float diff = neighbor_height - base_height;

            // Distance with stability clamp
            const float dist2 = static_cast<float>(dx * dx + dy * dy);
            const float distance = fmaxf(sqrtf(dist2), 1e-5f);

            // Inverse-square falloff (with +1 to avoid singularity)
            float weight = 1.0f / (distance * distance + 1.0f);

            if (diff > 0.0f) {
                occlusion += diff * weight;
            }
            total_weight += weight;
        }
    }

    // Normalize and invert to get AO
    float ao_value = 1.0f - (occlusion / fmaxf(total_weight, 1e-8f));
    // Clamp to [0,1]
    ao_value = fminf(fmaxf(ao_value, 0.0f), 1.0f);

    ao_map[base_index] = ao_value;
}

void ShaderMaps::generate_ao_map(
    const float *host_in, float *host_out,
    int width, int height,
    int radius, bool wrap) {



    printf("WARNING.... AO MAP BROKEN!\n");

    size_t in_size = width * height * sizeof(float);
    size_t out_size = width * height * sizeof(float);

    float *d_in = nullptr;
    float *d_out = nullptr;

    hipMalloc(&d_in, in_size);
    hipMalloc(&d_out, out_size);

    hipMemcpy(d_in, host_in, in_size, hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x,
              (height + block.y - 1) / block.y);

    generate_normal_map_kernel<<<grid, block>>>(
        d_in, d_out,
        width, height,
        radius, wrap);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel launch failed: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(host_out, d_out, out_size, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
}

#pragma endregion

} // namespace shader_maps_c