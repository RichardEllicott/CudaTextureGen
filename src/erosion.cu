#include "hip/hip_runtime.h"
#include "erosion.cuh"

namespace erosion {

#ifdef ENABLE_EROSION_TRIPWIRE
bool Erosion::instance_created = false; // set tripwire
#endif

#pragma region MAIN

#ifdef ENABLE_EROSION_JITTER
__global__ void initRand(hiprandState *states, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;
    hiprand_init(1234, idx, 0, &states[idx]);
}
#endif

__global__ void erode_kernel(
    Parameters *pars,
    int width, int height,

    float *heightmap, float *sediment

#ifdef ENABLE_EROSION_JITTER
    ,
    hiprandState *rand_states
#endif

) {

#ifdef ENABLE_EROSION_TILED_MEMORY
    __shared__ float tile[EROSION_BLOCK_SIZE + 2][EROSION_BLOCK_SIZE + 2]; // +2 for 1-cell border
#endif

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height)
        return;

    int idx = y * width + x;
    float h = heightmap[idx];
    float s = sediment[idx];

    // 8-way neighbor offsets
    int dx[8] = {-1, 1, 0, 0, -1, -1, 1, 1};
    int dy[8] = {0, 0, -1, 1, -1, 1, -1, 1};

    float total_slope = 0.0f;
    float slopes[8] = {0};

    // Compute slopes to neighbors
    for (int i = 0; i < 8; ++i) {

        int nx;
        int ny;

        if (pars->wrap) {
            nx = (x + dx[i] + width) % width;
            ny = (y + dy[i] + height) % height;
        } else {
            nx = x + dx[i];
            ny = y + dy[i];
            if (nx < 0 || nx >= width || ny < 0 || ny >= height)
                continue;
        }

        int nIdx = ny * width + nx;
        float nh = heightmap[nIdx];
        float slope = h - nh;

#ifdef ENABLE_EROSION_JITTER

        if (pars->jitter > 0.0f) {
            float rand = hiprand_uniform(&rand_states[idx]); // [0,1)
            slope += rand * pars->jitter;
        }

#endif

        if (slope > pars->slope_threshold) {
            slopes[i] = slope;
            total_slope += slope;
        }
    }

    // Erode and deposit based on slope
    float eroded = pars->erosion_rate * total_slope;
    h -= eroded;
    s += eroded;

    // Distribute sediment to neighbors
    for (int i = 0; i < 8; ++i) {
        if (slopes[i] > 0) {
            int nx = x + dx[i];
            int ny = y + dy[i];
            if (nx < 0 || nx >= width || ny < 0 || ny >= height)
                continue;

            int nIdx = ny * width + nx;
            float share = (slopes[i] / total_slope) * pars->deposition_rate * s;

            // Atomic to avoid race conditions
            atomicAdd(&heightmap[nIdx], share);
            atomicAdd(&sediment[nIdx], -share);
        }
    }

    // Write back
    heightmap[idx] = h;
    sediment[idx] = s;
}

#pragma endregion

#pragma region CLASS

// NOT REQUIRED
//         hipMemcpyToSymbol(HIP_SYMBOL(erosion::NAME), &p_##NAME, sizeof(TYPE)); \


void Erosion::run_erosion(float *host_data, int width, int height) {

    size_t size = width * height * sizeof(float);

#ifdef ENABLE_EROSION_JITTER
    hiprandState *dev_rand_states;
    CUDA_CHECK(
        hipMalloc(&dev_rand_states, width * height * sizeof(hiprandState)));
#endif

    // copy pars to gpu
    CUDA_CHECK(hipMalloc(&dev_pars, sizeof(Parameters)));
    CUDA_CHECK(hipMemcpy(dev_pars, &pars, sizeof(Parameters), hipMemcpyHostToDevice));

    // allocate memory
    CUDA_CHECK(hipMalloc(&dev_heightmap, size));
    CUDA_CHECK(hipMemcpy(dev_heightmap, host_data, size, hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&dev_water, size));
    CUDA_CHECK(hipMemset(dev_water, 0, size)); // start with no water

    // hipMalloc(&dev_outflow, size);
    // hipMemset(dev_outflow, 0, size);

    hipMalloc(&dev_sediment, size);
    hipMemset(dev_sediment, 0, size);

    dim3 block(pars.block_size, pars.block_size);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    // ⏲️ Timer
    auto start_time = std::chrono::high_resolution_clock::now();

#ifdef ENABLE_EROSION_JITTER
    initRand<<<grid, block>>>(dev_rand_states, width, height);
#endif

    // Loop on the host, but keep data on device
    for (int s = 0; s < pars.steps; ++s) {

        // rain_kernel<<<grid, block>>>(dev_water, width, height, rain_rate);
        // flow_kernel<<<grid, block>>>(dev_height, dev_water, dev_outflow, width, height);
        // erosion_kernel<<<grid, block>>>(dev_height, dev_water, dev_sediment, width, height, erosion_rate, deposition_rate);

        // sediment_transport_kernel<<<grid, block>>>(dev_height, dev_water, dev_sediment, width, height); // new

        // evaporation_kernel<<<grid, block>>>(dev_water, width, height, evaporation_rate);

        erode_kernel<<<grid, block>>>(
            dev_pars, width, height,
            dev_heightmap, dev_sediment

#ifdef ENABLE_EROSION_JITTER
            ,
            dev_rand_states
#endif
        );
    }
    hipDeviceSynchronize();

    // ⏲️ Timer
    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end_time - start_time;
    double seconds = elapsed.count();
    println("calculation time ⏱️: ", seconds * 1000.0, " ms");

    // Copy back once
    hipMemcpy(host_data, dev_heightmap, size, hipMemcpyDeviceToHost);

    // // free data
    CUDA_CHECK(hipFree(dev_pars));

    CUDA_CHECK(hipFree(dev_heightmap));
    CUDA_CHECK(hipFree(dev_water));
    CUDA_CHECK(hipFree(dev_sediment));
}

Erosion::Erosion() {

#ifdef ENABLE_EROSION_TRIPWIRE
    if (instance_created) {
        std::cerr << "ERROR: TerrainEroder already instantiated!" << std::endl;
        std::abort(); // or throw std::runtime_error
    }
    instance_created = true;
#endif
}

Erosion::~Erosion() {
#ifdef ENABLE_EROSION_TRIPWIRE
    instance_created = false;
#endif
}

#pragma endregion

} // namespace erosion