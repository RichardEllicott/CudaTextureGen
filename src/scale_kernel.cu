#include <hip/hip_runtime.h>

__global__ void scale_kernel(float* data, int n, float factor) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        data[idx] *= factor;
    }
}

extern "C" void scale_array(float* data, int n, float factor) {
    float* d_data;
    hipMalloc(&d_data, n * sizeof(float));
    hipMemcpy(d_data, data, n * sizeof(float), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    scale_kernel<<<blocks, threads>>>(d_data, n, factor);

    hipMemcpy(data, d_data, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_data);
}
