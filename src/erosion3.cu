#include "hip/hip_runtime.h"
#include "erosion3.cuh"
#include "noise_util.cuh"
#include <chrono>

#define OFFSET_ORDER 1 // 0 is the orginal WEIRD DOESSNT WORK, MIGHT BE THE OPPOSITE CODE, 1 modified
#define HASH_INT_ORDER

namespace TEMPLATE_NAMESPACE {


    // Apply inverse-square crater imprint onto a heightmap.
// h: heightmap (row-major), W,H: dimensions
// cx,cy: impact center in pixels (float for subpixel)
// k: excavation scale (meters per unit energy)
// r0: softening radius in pixels (prevents singularities)
// mask_radius: optional clamp for finite blast radius
__global__ void crater_imprint(float* h, int W, int H,
                               float cx, float cy,
                               float k, float r0, float mask_radius)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= W || y >= H) return;

    float dx = (x + 0.5f) - cx;
    float dy = (y + 0.5f) - cy;
    float r2 = dx*dx + dy*dy;

    if (mask_radius > 0.0f && r2 > mask_radius*mask_radius) return;

    float denom = r2 + r0*r0;        // softening
    float E = 1.0f / denom;          // inverse-square
    float dh = -k * E;               // excavation depth

    // Optional: taper center to avoid a pixel spike when r0 is small
    // dh *= (r2 / (r2 + r0*r0));

    int idx = y * W + x;
    h[idx] += dh;
}




__device__ __forceinline__ int wrap_or_clamp(int i, int n, bool wrap) {
    if (wrap) {
        int m = i % n;
        return m < 0 ? m + n : m;
    }
    return i < 0 ? 0 : (i >= n ? n - 1 : i);
}

// calculates the changes that need to occur
__global__ void flux_pass(
    const Parameters pars,
    int width, int height,
    const float *__restrict__ height_in,
    const float *__restrict__ water_in,
    const float *__restrict__ sediment_in,
    // outputs
    float *__restrict__ flux8, // 8 fluxes per cell (neighbor order)
    float *__restrict__ dh_out,
    float *__restrict__ ds_out,
    float *__restrict__ dw_out) {

    //
    //

#if OFFSET_ORDER == 0
    const float d_dist = 1.41421356;
    // const float d_dist = 1.0;
    const int2 offs[8] = {{-1, 0}, {1, 0}, {0, -1}, {0, 1}, {-1, -1}, {-1, 1}, {1, -1}, {1, 1}};
    // const float dist[8] = {1, 1, 1, 1, 1.41421356f, 1.41421356f, 1.41421356f, 1.41421356f};
    const float dist[8] = {1, 1, 1, 1, d_dist, d_dist, d_dist, d_dist};
#elif OFFSET_ORDER == 1
    const float d_dist = 1.41421356;
    // const float d_dist = 1.0;
    const int2 offs[8] = {{1, 0}, {0, -1}, {0, 1}, {-1, -1}, {-1, 1}, {1, -1}, {1, 1}, {-1, 0}};
    // const float dist[8] = {1, 1, 1, 1, 1.41421356f, 1.41421356f, 1.41421356f, 1.41421356f};
    const float dist[8] = {1, 1, 1, d_dist, d_dist, d_dist, d_dist, 1};
#endif

    //
    //

    //
    //
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height)
        return;

    int idx = y * width + x;
    float h = height_in[idx];
    float w = water_in[idx] + pars.rain_rate;
    float s_cur = sediment_in[idx];

    float slopes[8];
    float sum_slope = 0.f;

    // neighbor heights and slopes
    for (int n = 0; n < 8; ++n) {

#ifdef HASH_INT_ORDER
        auto i2 = (n + noise_util::hash_int(x, y, 0)) % 8;
#else
        auto i2 = n;
#endif

        int nx = wrap_or_clamp(x + offs[i2].x, width, pars.wrap);
        int ny = wrap_or_clamp(y + offs[i2].y, height, pars.wrap);
        int nidx = ny * width + nx;
        float nh = height_in[nidx];
        float s = (h - nh) / dist[i2];
        float sd = s > 0.f ? s : 0.f;
        slopes[i2] = sd;
        sum_slope += sd;
    }

    float outflow_cap = fminf(w, pars.w_max);
    float outflow_sum = 0.f;

    // proportional flux
    float *cell_flux = &flux8[idx * 8];
    if (sum_slope > 1e-6f && outflow_cap > 0.f) {
        for (int i = 0; i < 8; ++i) {
            float q = (slopes[i] / sum_slope) * outflow_cap;
            cell_flux[i] = q;
            outflow_sum += q;
        }
    } else {
        for (int i = 0; i < 8; ++i)
            cell_flux[i] = 0.f;
    }

    // velocity proxy and capacity
    float v = 0.f;
    for (int i = 0; i < 8; ++i)
        v += cell_flux[i] * slopes[i];
    float C = pars.capacity * v;

    float erode = 0.f, deposit = 0.f;
    if (C > s_cur) {
        erode = pars.erode * (C - s_cur);
    } else {
        deposit = pars.deposit * (s_cur - C);
    }

    // write deltas (applied in pass B)
    dh_out[idx] = deposit - erode;              // positive = deposition raises height
    ds_out[idx] = erode - deposit;              // sediment increases when eroding, decreases when depositing
    dw_out[idx] = -outflow_sum - pars.evap * w; // water loss: outflow + evaporation
}

// apply the changes
__global__ void apply_pass(
    const Parameters pars,
    int width, int height,
    const float *__restrict__ water_in,
    const float *__restrict__ sediment_in,
    const float *__restrict__ height_in,
    const float *__restrict__ flux, // 8 fluxes per cell
    const float *__restrict__ dh,   // erosion/deposition delta
    const float *__restrict__ ds,   // sediment delta
    const float *__restrict__ dw,   // water delta (loss)
    float *__restrict__ water_out,
    float *__restrict__ sediment_out,
    float *__restrict__ height_out) {

#if OFFSET_ORDER == 0
    const int2 offs[8] = {{-1, 0}, {1, 0}, {0, -1}, {0, 1}, {-1, -1}, {-1, 1}, {1, -1}, {1, 1}};
#elif OFFSET_ORDER == 1
    const int2 offs[8] = {{1, 0}, {0, -1}, {0, 1}, {-1, -1}, {-1, 1}, {1, -1}, {1, 1}, {-1, 0}};
#endif

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height)
        return;

    int idx = y * width + x;

    float w = water_in[idx];
    float s = sediment_in[idx];
    float h = height_in[idx];

    // apply local deltas
    w += dw[idx];
    s += ds[idx];
    h += dh[idx];

    // add incoming flux from neighbors
    float inflow = 0.f;

    // #define MODIFY_THIS

#ifndef MODIFY_THIS
#else
    float inflow = 0.f;
#endif

    for (int n = 0; n < 8; ++n) {

#ifdef HASH_INT_ORDER
        auto i2 = (n + noise_util::hash_int(x, y, 0)) % 8;
#else
        auto i2 = n;
#endif

        int nx = x + offs[i2].x;
        int ny = y + offs[i2].y;

        // if (nx < 0 || nx >= width || ny < 0 || ny >= height)
        // continue;

        nx = wrap_or_clamp(x + offs[i2].x, width, pars.wrap); // note we lost continue
        ny = wrap_or_clamp(y + offs[i2].y, height, pars.wrap);

        int nidx = ny * width + nx;
        // opposite direction index (neighbor sending to me)

#ifndef MODIFY_THIS
        int opp = i2 ^ 1; // crude: 0<->1, 2<->3, 4<->6, 5<->7   // TRYING TO FIND OPPOSITE
        inflow += flux[nidx * 8 + opp];
#else
        int opp_i = opp[n];
        float q_in = flux[nidx * 8 + opp_i];
        float conc = sediment_in[nidx] / fmaxf(water_in[nidx], 1e-6f);
        inflow_s += q_in * conc;

#endif
    }
    w += inflow;

    // write out
    water_out[idx] = fmaxf(0.f, w);
    sediment_out[idx] = fmaxf(0.f, s);
    height_out[idx] = fmaxf(0.f, h);
}

// recomended stuff

// water_in.resize(...);
// water_out.resize(...);
// sediment_in.resize(...);
// sediment_out.resize(...);
// height_in.resize(...);
// height_out.resize(...);

// flux0.resize(width, height, 8); // 8 neighbors per cell
// dh_out.resize(...);
// ds_out.resize(...);
// dw_out.resize(...);

// pars.rain_rate;
// pars.evap_rate;
// pars.w_max;
// pars.k_capacity;
// pars.k_erode;
// pars.k_deposit;
// pars.wrap;
// pars.epsilon;

//
//
//
//

// we are currently talking about erosion here
// https://copilot.microsoft.com/chats/fEfp39jA1SjQeW7yA4D4Y

// but we need more advanced frameworking

void TEMPLATE_CLASS_NAME::process() {

    printf("<<< Erosion Process >>>\n");

    height_map.upload();

    pars.width = height_map.get_width();
    pars.height = height_map.get_height();

    water_map.resize(pars.width, pars.height);
    sediment_map.resize(pars.width, pars.height);
    // flux8.resize(pars.width, pars.height);
    dh_out.resize(pars.width, pars.height);
    ds_out.resize(pars.width, pars.height);
    dw_out.resize(pars.width, pars.height);

    water_map.clear();
    sediment_map.clear();
    // flux8.clear();
    dh_out.clear();
    ds_out.clear();
    dw_out.clear();

    water_map.upload();
    sediment_map.upload();
    // flux8.upload();
    dh_out.upload();
    ds_out.upload();
    dw_out.upload();

    size_t array_size = pars.width * pars.height;

    // allocate a flux map 8x larger than the other maps
    core::CudaArrayManager<float> flux8;
    flux8.resize(array_size * 8);
    flux8.zero_device();

    //
    //

    // out maps, will be freed when we go out of scope
    core::CudaArrayManager<float> height_map_out;
    height_map_out.resize(array_size);
    height_map_out.zero_device();

    core::CudaArrayManager<float> water_map_out;
    water_map_out.resize(array_size);
    water_map_out.zero_device();

    core::CudaArrayManager<float> sediment_map_out;
    sediment_map_out.resize(array_size);
    sediment_map_out.zero_device();

    //
    //
    // core::CudaStruct<Parameters> gpu_pars(pars); // automaticly uploads and free

    // #define X(TYPE, NAME) \
    //     NAME.upload_to_device();
    //     TEMPLATE_CLASS_MAPS
    // #undef X

    core::CudaStruct<Parameters> gpu_pars(pars); // automaticly uploads and free

    dim3 block(pars._block, pars._block);
    dim3 grid((pars.width + block.x - 1) / block.x,
              (pars.height + block.y - 1) / block.y);


                  auto start_time = std::chrono::high_resolution_clock::now(); // ⏲️ Timer


    for (int i = 0; i < pars.steps; i += 2) {
        // --- timestep 1: in -> out ---
        flux_pass<<<grid, block>>>(
            pars,
            pars.width, pars.height,
            height_map.dev_ptr(),
            water_map.dev_ptr(),
            sediment_map.dev_ptr(),
            flux8.data(),
            dh_out.dev_ptr(),
            ds_out.dev_ptr(),
            dw_out.dev_ptr());

        apply_pass<<<grid, block>>>(
            pars,
            pars.width, pars.height,
            height_map.dev_ptr(),
            water_map.dev_ptr(),
            sediment_map.dev_ptr(),
            flux8.data(),
            dh_out.dev_ptr(),
            ds_out.dev_ptr(),
            dw_out.dev_ptr(),
            height_map_out.data(),
            water_map_out.data(),
            sediment_map_out.data());

        // --- timestep 2: out -> in ---
        flux_pass<<<grid, block>>>(
            pars,
            pars.width, pars.height,
            height_map_out.data(),
            water_map_out.data(),
            sediment_map_out.data(),
            flux8.data(),
            dh_out.dev_ptr(),
            ds_out.dev_ptr(),
            dw_out.dev_ptr());

        apply_pass<<<grid, block>>>(
            pars,
            pars.width, pars.height,
            height_map_out.data(),
            water_map_out.data(),
            sediment_map_out.data(),
            flux8.data(),
            dh_out.dev_ptr(),
            ds_out.dev_ptr(),
            dw_out.dev_ptr(),
            height_map.dev_ptr(),
            water_map.dev_ptr(),
            sediment_map.dev_ptr());
    }

    auto end_time = std::chrono::high_resolution_clock::now(); // ⏲️ Timer
    std::chrono::duration<double> elapsed = end_time - start_time;
    double seconds = elapsed.count();
    printf("calculation time: %.2f ms\n", seconds * 1000.0); // ⏱️

    height_map.download();
    water_map.download();
    sediment_map.download();

    flux8.free_device();

    // free all maps
#define X(TYPE, NAME)   \
    NAME.free_device(); \
    TEMPLATE_CLASS_MAPS
#undef X
}

} // namespace TEMPLATE_NAMESPACE
